#include "hip/hip_runtime.h"

#include "kernels.cuh"

int currDevice = -1;
hipDeviceProp_t prop;
int numThreads = 256;

__global__ void fillZeros(float *buf, size_t size)
{
	const size_t numThreads = blockDim.x * gridDim.x;
	const size_t threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (size_t i = threadID; i < size; i += numThreads)
	{
		buf[i] = 0.0f;
	}
}
void fillWithZeroesKernel(float *buf, int size, hipStream_t s)
{

	// TODO: Implement this when there are several GPUs/
	// Assuming one GPU for now
	// This code is a scalability measure that
	// ensures that the kernel can fit on the GPU
	// since the histogram can get very large
	/*if (currDevice == -1) {
		checkCudaErrors(hipGetDevice(&currDevice));
		hipGetDeviceProperties(&prop, currDevice);
		int maxGridSize[3] = { prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2] };

	}*/
	/*
	int device;
	checkCudaErrors(hipGetDevice(&device));
	if (currDevice == -1 || currDevice != device) {
		currDevice = device;
		hipGetDeviceProperties(&prop, device);
	}*/

	int numBlocks = (size + numThreads - 1) / numThreads;
	if (numBlocks > 1024) {
		int num_iterations = (numBlocks + 1023) / 1024;
		for (int i = 0; i < num_iterations; i++) {
			int curr_block_size = numBlocks > 1024 ? 1024 : numBlocks;
			int curr_size = curr_block_size == 1024 ? curr_block_size * numThreads : size;
			if (s)
			{
				fillZeros << <numThreads, curr_block_size, 0, s >> > (buf, curr_size);
			}
			else
			{
				fillZeros << <numThreads, curr_block_size >> > (buf, curr_size);
			}
			buf += curr_block_size * numThreads;
			size -= curr_size;
			numBlocks -= curr_block_size;
		}
		
	}
	else {
		if (s)
		{
			fillZeros << <numThreads, numBlocks, 0, s >> > (buf, size);
		}
		else
		{
			fillZeros << <numThreads, numBlocks >> > (buf, size);
		}
	}
	
	getLastCudaError("Kernel Launch Failure");
}